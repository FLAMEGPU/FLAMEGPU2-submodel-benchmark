#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <array>

#include "flamegpu/flamegpu.h"

// Agent state variables
#define AGENT_STATUS_UNOCCUPIED 0
#define AGENT_STATUS_OCCUPIED 1
#define AGENT_STATUS_MOVEMENT_REQUESTED 2
#define AGENT_STATUS_MOVEMENT_UNRESOLVED 3

// Growback variables
#define ENV_SUGAR_GROWBACK_RATE 1
#define ENV_SUGAR_MAX_CAPACITY 4
#define MIN_HOTSPOT_DISTANCE 20

#define MIN_INIT_AGENT_SUAGR_WEALTH 5
#define MAX_INIT_AGENT_SUAGR_WEALTH 25

#define MIN_INIT_METABOLISM 1
#define MAX_INIT_METABOLISM 4
#define PROBABILITY_OF_OCCUPATION 0.17f


// Visualisation mode (0=occupied/move status, 1=occupied/sugar/level)
#define VIS_MODE 1

// Number of steps and repetitions for different expeirments.
#define VIS_STEPS 0
#define VIS_SEED 0
#define VIS_SIMULATION_SPEED 2  // Target visualistaion speed

#define BENCHMARK_REPETITIONS 3
#define BENCHMARK_STEPS 100
#define BENCHMARK_REPETITIONS 3
#define BENCHMARK_SEED 0

// Counter for number of iterations required to resolve conflicting movements
unsigned int exit_condition_iterations = 0;
unsigned int occupied = 0;
std::array<unsigned int, 9> step_unresolved_count;
std::array<unsigned int, 9> mean_unresolved_count;


FLAMEGPU_AGENT_FUNCTION(metabolise_and_growback, flamegpu::MessageNone, flamegpu::MessageNone) {
    int sugar_level = FLAMEGPU->getVariable<int>("sugar_level");
    int env_sugar_level = FLAMEGPU->getVariable<int>("env_sugar_level");
    int env_max_sugar_level = FLAMEGPU->getVariable<int>("env_max_sugar_level");
    int status = FLAMEGPU->getVariable<int>("status");
    // metabolise if occupied
    if (status == AGENT_STATUS_OCCUPIED || status == AGENT_STATUS_MOVEMENT_UNRESOLVED) {
        // store any sugar present in the cell
        if (env_sugar_level > 0) {
            sugar_level += env_sugar_level;
            // Occupied cells are marked as -1 sugar.
            env_sugar_level = -1;
        }

        // metabolise
        sugar_level -= FLAMEGPU->getVariable<int>("metabolism");

        // check if agent dies
        if (sugar_level == 0) {
            status = AGENT_STATUS_UNOCCUPIED;
            FLAMEGPU->setVariable<int>("agent_id", -1);
            env_sugar_level = 0;
            FLAMEGPU->setVariable<int>("metabolism", 0);
        }
    }

    // growback if unoccupied
    if (status == AGENT_STATUS_UNOCCUPIED) {
        env_sugar_level += ENV_SUGAR_GROWBACK_RATE;
        if (env_sugar_level > env_max_sugar_level) {
            env_sugar_level = env_max_sugar_level;
        }
    }

    // set all active agents to unresolved as they may now want to move
    if (status == AGENT_STATUS_OCCUPIED) {
        status = AGENT_STATUS_MOVEMENT_UNRESOLVED;
    }
    FLAMEGPU->setVariable<int>("sugar_level", sugar_level);
    FLAMEGPU->setVariable<int>("env_sugar_level", env_sugar_level);
    FLAMEGPU->setVariable<int>("status", status);

    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(output_cell_status, flamegpu::MessageNone, flamegpu::MessageArray2D) {
    unsigned int agent_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    unsigned int agent_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);
    FLAMEGPU->message_out.setVariable("location_id", FLAMEGPU->getID());
    FLAMEGPU->message_out.setVariable("status", FLAMEGPU->getVariable<int>("status"));
    FLAMEGPU->message_out.setVariable("env_sugar_level", FLAMEGPU->getVariable<int>("env_sugar_level"));
    FLAMEGPU->message_out.setIndex(agent_x, agent_y);
    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(movement_request, flamegpu::MessageArray2D, flamegpu::MessageArray2D) {
    int best_sugar_level = -1;
    float best_sugar_random = -1;
    flamegpu::id_t best_location_id = flamegpu::ID_NOT_SET;

    // if occupied then look for empty cells {
    // find the best location to move to (ensure we don't just pick first cell with max value)
    int status = FLAMEGPU->getVariable<int>("status");

    unsigned int agent_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    unsigned int agent_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    // if occupied then look for empty cells
    if (status == AGENT_STATUS_MOVEMENT_UNRESOLVED) {
        for (auto current_message : FLAMEGPU->message_in.wrap(agent_x, agent_y)) {
            // if location is unoccupied then check for empty locations
            if (current_message.getVariable<int>("status") == AGENT_STATUS_UNOCCUPIED) {
                // if the sugar level at current location is better than currently stored then update
                int message_env_sugar_level = current_message.getVariable<int>("env_sugar_level");
                float message_priority = FLAMEGPU->random.uniform<float>();
                if ((message_env_sugar_level > best_sugar_level) ||
                    (message_env_sugar_level == best_sugar_level && message_priority > best_sugar_random)) {
                    best_sugar_level = message_env_sugar_level;
                    best_sugar_random = message_priority;
                    best_location_id = current_message.getVariable<flamegpu::id_t>("location_id");
                }
            }
        }

        // if the agent has found a better location to move to then update its state
        // if there is a better location to move to then state indicates a movement request
        status = best_location_id != flamegpu::ID_NOT_SET ? AGENT_STATUS_MOVEMENT_REQUESTED : AGENT_STATUS_OCCUPIED;
        FLAMEGPU->setVariable<int>("status", status);
    }

    // add a movement request
    FLAMEGPU->message_out.setVariable<int>("agent_id", FLAMEGPU->getVariable<int>("agent_id"));
    FLAMEGPU->message_out.setVariable<flamegpu::id_t>("location_id", best_location_id);
    FLAMEGPU->message_out.setVariable<int>("sugar_level", FLAMEGPU->getVariable<int>("sugar_level"));
    FLAMEGPU->message_out.setVariable<int>("metabolism", FLAMEGPU->getVariable<int>("metabolism"));
    FLAMEGPU->message_out.setIndex(agent_x, agent_y);

    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(movement_response, flamegpu::MessageArray2D, flamegpu::MessageArray2D) {
    int best_request_id = -1;
    float best_request_priority = -1;
    int best_request_sugar_level = -1;
    int best_request_metabolism = -1;

    int status = FLAMEGPU->getVariable<int>("status");
    const flamegpu::id_t location_id = FLAMEGPU->getID();
    const unsigned int agent_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    const unsigned int agent_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    for (auto current_message : FLAMEGPU->message_in.wrap(agent_x, agent_y)) {
        // if the location is unoccupied then check for agents requesting to move here
        if (status == AGENT_STATUS_UNOCCUPIED) {
            // check if request is to move to this location
            if (current_message.getVariable<flamegpu::id_t>("location_id") == location_id) {
                // check the priority and maintain the best ranked agent
                float message_priority = FLAMEGPU->random.uniform<float>();
                if (message_priority > best_request_priority) {
                    best_request_id = current_message.getVariable<int>("agent_id");
                    best_request_priority = message_priority;
                }
            }
        }
    }

    // if the location is unoccupied and an agent wants to move here then do so and send a response
    if ((status == AGENT_STATUS_UNOCCUPIED) && (best_request_id >= 0)) {
        FLAMEGPU->setVariable<int>("status", AGENT_STATUS_OCCUPIED);
        // move the agent to here and consume the cell's sugar
        best_request_sugar_level += FLAMEGPU->getVariable<int>("env_sugar_level");
        FLAMEGPU->setVariable<int>("agent_id", best_request_id);
        FLAMEGPU->setVariable<int>("sugar_level", best_request_sugar_level);
        FLAMEGPU->setVariable<int>("metabolism", best_request_metabolism);
        FLAMEGPU->setVariable<int>("env_sugar_level", -1);
    }

    // add a movement response
    FLAMEGPU->message_out.setVariable<int>("agent_id", best_request_id);
    FLAMEGPU->message_out.setIndex(agent_x, agent_y);

    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(movement_transaction, flamegpu::MessageArray2D, flamegpu::MessageNone) {
    int status = FLAMEGPU->getVariable<int>("status");
    int agent_id = FLAMEGPU->getVariable<int>("agent_id");
    unsigned int agent_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    unsigned int agent_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    for (auto current_message : FLAMEGPU->message_in.wrap(agent_x, agent_y)) {
        // if location contains an agent wanting to move then look for responses allowing relocation
        if (status == AGENT_STATUS_MOVEMENT_REQUESTED) {  // if the movement response request came from this location
            if (current_message.getVariable<int>("agent_id") == agent_id) {
                // remove the agent and reset agent specific variables as it has now moved
                status = AGENT_STATUS_UNOCCUPIED;
                FLAMEGPU->setVariable<int>("agent_id", -1);
                FLAMEGPU->setVariable<int>("sugar_level", 0);
                FLAMEGPU->setVariable<int>("metabolism", 0);
                FLAMEGPU->setVariable<int>("env_sugar_level", 0);
            }
        }
    }

    // if request has not been responded to then agent is unresolved
    if (status == AGENT_STATUS_MOVEMENT_REQUESTED) {
        status = AGENT_STATUS_MOVEMENT_UNRESOLVED;
    }

    FLAMEGPU->setVariable<int>("status", status);

    return flamegpu::ALIVE;
}


FLAMEGPU_EXIT_CONDITION(MovementExitCondition) {
    // static unsigned int exit_condition_iterations = 0;
    exit_condition_iterations++;

    // Max iterations 9
    if (exit_condition_iterations < 9) {
        // Agent movements still unresolved
        unsigned int unresolved = FLAMEGPU->agent("agent").count("status", AGENT_STATUS_MOVEMENT_UNRESOLVED);
        step_unresolved_count[exit_condition_iterations] = unresolved;
        if (unresolved) {
            return flamegpu::CONTINUE;
        }
    }

    //save the number of occupied cells after exit condition
    occupied = FLAMEGPU->agent("agent").count("status", AGENT_STATUS_OCCUPIED);

    // exit_condition_iterations = 0;
    return flamegpu::EXIT;
}


/**
 * Construct the common components of agent shared between both parent and submodel
 */
flamegpu::AgentDescription makeCoreAgent(flamegpu::ModelDescription &model) {
    flamegpu::AgentDescription  agent = model.newAgent("agent");
    agent.newVariable<unsigned int, 2>("pos");
    agent.newVariable<int>("agent_id");
    agent.newVariable<int>("status");
    // agent specific variables
    agent.newVariable<int>("sugar_level");
    agent.newVariable<int>("metabolism");
    // environment specific var
    agent.newVariable<int>("env_sugar_level");
    agent.newVariable<int>("env_max_sugar_level");
#ifdef FLAMEGPU_VISUALISATION
    // Redundant seperate floating point position vars for vis
    agent.newVariable<float>("x");
    agent.newVariable<float>("y");
#endif
    return agent;
}


bool hotspot_distance_check(const std::vector<std::array<unsigned int, 2>>& sugar_hotspots, const std::array<unsigned int, 2>& hs, const unsigned int gridWidth) {
    bool pass = true;
    for (auto& h : sugar_hotspots) {
        // calculate the distance between spots
        unsigned int dx = std::abs(static_cast<int>(std::get<0>(hs)) - static_cast<int>(std::get<0>(h)));
        unsigned int dy = std::abs(static_cast<int>(std::get<1>(hs)) - static_cast<int>(std::get<1>(h)));
        // if distance in a dimension is greater than half grid width then points are closer via wrapping
        if (dx > (gridWidth >>1))
            dx = gridWidth - dx;
        if (dy > (gridWidth >> 1))
            dy = gridWidth - dy;

        if (std::sqrt((pow(dx, 2.0f) + pow(dy, 2.0f))) < static_cast<float>(MIN_HOTSPOT_DISTANCE) )
            pass = false;
    }
    return pass;
}


typedef struct Experiment {
    Experiment(std::string title,
        unsigned int initialGridWidth, unsigned int finalGridWidth, unsigned int gridWidthIncrement, std::vector<float> p_occupations,
        unsigned int repetitions,
        unsigned int steps,
        bool histogram ) {
        this->title = title;
        this->initialGridWidth = initialGridWidth;
        this->finalGridWidth = finalGridWidth;
        this->gridWidthIncrement = gridWidthIncrement;
        this->p_occupations = p_occupations;
        this->repetitions = repetitions;
        this->steps = steps;
        this->histogram = histogram;
    }
    std::string title;
    unsigned int initialGridWidth, finalGridWidth, gridWidthIncrement;
    std::vector<float> p_occupations;
    unsigned int repetitions;
    unsigned int steps;
    bool histogram;
} Experiment;


int main(int argc, const char** argv) {
    std::vector<Experiment> experiments;
    // For visualisation define only a single execution othger wise describe benchmark experiment
#ifdef VISUALISATION
    Experiment visualisationExperiment("visualisation", 256, 256, 256, std::vector<float>({PROBABILITY_OF_OCCUPATION}), BENCHMARK_REPETITIONS, VIS_STEPS, false);
    experiments.push_back(visualisationExperiment);
#else
    // Performacne scaling experiment to recoprd performance with increase in model size
    Experiment performance_scaling("performance_scaling", 256, 4096, 256, std::vector<float>({PROBABILITY_OF_OCCUPATION}), BENCHMARK_REPETITIONS, BENCHMARK_STEPS, false);
    experiments.push_back(performance_scaling);

    // Performacne sweep of occupation experiment
    Experiment occupation("occupation", 512, 512, 512, std::vector<float>({ 0.02f,0.04f,0.08f,0.16f,0.32f,0.64f }), BENCHMARK_REPETITIONS, BENCHMARK_STEPS, false);
    experiments.push_back(occupation);

    // Perform resolution experiment
    Experiment resolution_steps("resolution_steps", 512, 512, 512, std::vector<float>({ 0.02f,0.04f,0.08f,0.16f,0.32f,0.64f }), BENCHMARK_REPETITIONS, 3, true);
    experiments.push_back(resolution_steps);

#endif

    for (Experiment experiment : experiments) {
        std::cout << std::endl << "Starting experiment: " << experiment.title << std::endl;

        // Pandas logging
        std::string csvFileName = "" + experiment.title + ".csv";
        std::ofstream csv(csvFileName);
        std::string csvFileNameStep = "" + experiment.title + "perStep.csv";
        std::ofstream csv_step(csvFileNameStep);

        if (experiment.histogram) {
            csv << "repetition,grid_width,pop_size,p_occupation,mean_pop_count,resolution_iterations,mean_unresolved_count" << std::endl;
        } else {
            csv << "repetition,grid_width,pop_size,p_occupation,s_step_mean,pop_count_mean" << std::endl;
            csv_step << "repetition,grid_width,pop_size,p_occupation,step,s_step,pop_count" << std::endl;
        }

        // number of repitions of experiment
        for (unsigned int repetition = 0; repetition < experiment.repetitions; repetition++) {
            // increment grid width
            for (unsigned int gridWidth = experiment.initialGridWidth; gridWidth <= experiment.finalGridWidth; gridWidth += experiment.gridWidthIncrement) {
                // increment probabiolity of occupations
                for (float& pOccupation : experiment.p_occupations) {
                    unsigned int popSize = gridWidth * gridWidth;

                    std::cout << "Staring run with popSize: " << popSize << ", gridthWidth: " << gridWidth << " proabilityOccupation:" << pOccupation << std::endl;

                    flamegpu::ModelDescription submodel("Movement_model");
                    {  // Define sub model for conflict resolution
                        /**
                         * Messages
                         */
                        {   // cell_status message
                            flamegpu::MessageArray2D::Description message = submodel.newMessage<flamegpu::MessageArray2D>("cell_status");
                            message.newVariable<flamegpu::id_t>("location_id");
                            message.newVariable<int>("status");
                            message.newVariable<int>("env_sugar_level");
                            message.setDimensions(gridWidth, gridWidth);
                        }
                        {   // movement_request message
                            flamegpu::MessageArray2D::Description message = submodel.newMessage<flamegpu::MessageArray2D>("movement_request");
                            message.newVariable<int>("agent_id");
                            message.newVariable<flamegpu::id_t>("location_id");
                            message.newVariable<int>("sugar_level");
                            message.newVariable<int>("metabolism");
                            message.setDimensions(gridWidth, gridWidth);
                        }
                        {   // movement_response message
                            flamegpu::MessageArray2D::Description message = submodel.newMessage<flamegpu::MessageArray2D>("movement_response");
                            message.newVariable<flamegpu::id_t>("location_id");
                            message.newVariable<int>("agent_id");
                            message.setDimensions(gridWidth, gridWidth);
                        }
                        /**
                         * Agents
                         */
                        {
                            flamegpu::AgentDescription agent = makeCoreAgent(submodel);
                            auto fn_output_cell_status = agent.newFunction("output_cell_status", output_cell_status);
                            {
                                fn_output_cell_status.setMessageOutput("cell_status");
                            }
                            auto fn_movement_request = agent.newFunction("movement_request", movement_request);
                            {
                                fn_movement_request.setMessageInput("cell_status");
                                fn_movement_request.setMessageOutput("movement_request");
                            }
                            auto fn_movement_response = agent.newFunction("movement_response", movement_response);
                            {
                                fn_movement_response.setMessageInput("movement_request");
                                fn_movement_response.setMessageOutput("movement_response");
                            }
                            auto fn_movement_transaction = agent.newFunction("movement_transaction", movement_transaction);
                            {
                                fn_movement_transaction.setMessageInput("movement_response");
                            }
                        }

                        /**
                         * Globals
                         */
                        {
                            // flamegpu::EnvironmentDescription  &env = model.Environment();
                        }

                        /**
                         * Control flow
                         */
                        {   // Layer #1
                            flamegpu::LayerDescription layer = submodel.newLayer();
                            layer.addAgentFunction(output_cell_status);
                        }
                        {   // Layer #2
                            flamegpu::LayerDescription layer = submodel.newLayer();
                            layer.addAgentFunction(movement_request);
                        }
                        {   // Layer #3
                            flamegpu::LayerDescription layer = submodel.newLayer();
                            layer.addAgentFunction(movement_response);
                        }
                        {   // Layer #4
                            flamegpu::LayerDescription layer = submodel.newLayer();
                            layer.addAgentFunction(movement_transaction);
                        }
                        submodel.addExitCondition(MovementExitCondition);
                    }

                    flamegpu::ModelDescription model("submodel-benchmark");

                    /**
                     * Agents
                     */
                    {   // Per cell agent
                        flamegpu::AgentDescription agent = makeCoreAgent(model);
                        // Functions
                        agent.newFunction("metabolise_and_growback", metabolise_and_growback);
                    }

                    /**
                     * Submodels
                     */
                    flamegpu::SubModelDescription movement_sub = model.newSubModel("movement_conflict_resolution_model", submodel);
                    {
                        movement_sub.bindAgent("agent", "agent", true, true);
                    }

                    /**
                     * Globals
                     */
                    {
                        // flamegpu::EnvironmentDescription  &env = model.Environment();
                    }

                    /**
                     * Control flow
                     */
                    {   // Layer #1
                        flamegpu::LayerDescription layer = model.newLayer();
                        layer.addAgentFunction(metabolise_and_growback);
                    }
                    {   // Layer #2
                        flamegpu::LayerDescription layer = model.newLayer();
                        layer.addSubModel(movement_sub);
                    }

                    /**
                     * Create Model Runner
                     */
                    flamegpu::CUDASimulation  cudaSimulation(model);


                    /**
                     * Create visualisation
                     * @note FLAMEGPU2 doesn't currently have proper support for discrete/2d visualisations
                     */
#ifdef VISUALISATION
                    flamegpu::visualiser::ModelVis& visualisation = cudaSimulation.getVisualisation();
                    {
                        visualisation.setSimulationSpeed(VIS_SIMULATION_SPEED);
                        visualisation.setInitialCameraLocation(gridWidth / 2.0f, gridWidth / 2.0f, 225.0f);
                        visualisation.setInitialCameraTarget(gridWidth / 2.0f, gridWidth / 2.0f, 0.0f);
                        visualisation.setCameraSpeed(0.001f * gridWidth);
                        visualisation.setViewClips(0.1f, 5000);
                        visualisation.setClearColor(1.0f, 1.0f, 1.0f);
                        visualisation.setFPSColor(0.0f, 0.0f, 0.0f);
                        visualisation.setBeginPaused(true);
                        auto& agt = visualisation.addAgent("agent");
                        // Position vars are named x, y, z; so they are used by default
                        agt.setModel(flamegpu::visualiser::Stock::Models::CUBE);  // 5 unwanted faces!
                        agt.setModelScale(1.0f);
#if VIS_MODE == 0
                        flamegpu::visualiser::DiscreteColor<int> cell_colors = flamegpu::visualiser::DiscreteColor<int>("status", flamegpu::visualiser::Color{ "#666" });
                        cell_colors[AGENT_STATUS_UNOCCUPIED] = flamegpu::visualiser::Stock::Colors::RED;
                        cell_colors[AGENT_STATUS_OCCUPIED] = flamegpu::visualiser::Stock::Colors::GREEN;
                        cell_colors[AGENT_STATUS_MOVEMENT_REQUESTED] = flamegpu::visualiser::Stock::Colors::BLUE;  // Not possible, only occurs inside the submodel
                        cell_colors[AGENT_STATUS_MOVEMENT_UNRESOLVED] = flamegpu::visualiser::Stock::Colors::WHITE;
                        agt.setColor(cell_colors);
#else
                        flamegpu::visualiser::DiscreteColor<int> cell_colors = flamegpu::visualiser::DiscreteColor<int>("env_sugar_level", flamegpu::visualiser::Stock::Palettes::Viridis(ENV_SUGAR_MAX_CAPACITY + 1), flamegpu::visualiser::Color{ "#f00" });
                        agt.setColor(cell_colors);
#endif
                    }
                    visualisation.activate();
#endif

                    /**
                     * Initialisation
                     */
                    cudaSimulation.initialise(argc, argv);
                    // set (override) the number of steps
#ifdef VISUALISATION
                    cudaSimulation.SimulationConfig().steps = 0;
#else
                    cudaSimulation.SimulationConfig().steps = BENCHMARK_STEPS;
#endif
                    // Set the seed for the simualtion. This must be done after CLI parsing as it is not currently possible to set a CLI over-rideable default for the vis mode.
#ifdef VISUALISATION
                    cudaSimulation.SimulationConfig().random_seed = VIS_SEED;
#else
                    cudaSimulation.SimulationConfig().random_seed = BENCHMARK_SEED + repetition;
#endif
                    // Apply the configuration, to ensure that the seed will be used as intended.
                    cudaSimulation.applyConfig();
                    if (cudaSimulation.getSimulationConfig().input_file.empty()) {
                        std::mt19937_64 rng(cudaSimulation.getSimulationConfig().random_seed);
                        // Pre init, decide the sugar hotspots
                        std::vector<std::array<unsigned int, 2>> sugar_hotspots;
                        {
                            std::uniform_int_distribution<unsigned int> width_dist(0, gridWidth - 1);
                            std::uniform_int_distribution<unsigned int> height_dist(0, gridWidth - 1);
                            // There are a number of hotspots which create an average denisty based on that of the original model
                            unsigned int num_hotspots = (2 * gridWidth * gridWidth) / (49 * 49);
                            for (unsigned int h = 0; h < num_hotspots; h++) {
                                // create random position for new hotspot
                                std::array<unsigned int, 2> hs = { width_dist(rng), height_dist(rng) };
                                // recursively ensure that the a random position is not within an euclidean distance of 10
                                unsigned int attempts = 0;
                                while (!hotspot_distance_check(sugar_hotspots, hs, gridWidth)) {
                                    hs = { width_dist(rng), height_dist(rng) };
                                    attempts++;
                                    // give up if no position found after 100 attempts
                                    if (attempts == 100) {
                                        std::cout << "Warning: Maximum attempts reached creating unique location for sugar hotspot." << std::endl;
                                        break;
                                    }
                                }
                                // add hostpot after it has passed the distance checks
                                sugar_hotspots.push_back(hs);
                            }
                        }


                        // Currently population has not been init, so generate an agent population on the fly
                        const unsigned int CELL_COUNT = gridWidth * gridWidth;
                        std::uniform_real_distribution<float> normal(0, 1);
                        std::uniform_int_distribution<int> agent_sugar_dist(MIN_INIT_AGENT_SUAGR_WEALTH, MAX_INIT_AGENT_SUAGR_WEALTH);
                        std::uniform_int_distribution<int> agent_metabolism_dist(MIN_INIT_METABOLISM, MAX_INIT_METABOLISM);
                        unsigned int i = 0;
                        unsigned int agent_id = 0;
                        flamegpu::AgentVector init_pop(model.Agent("agent"), CELL_COUNT);
                        for (unsigned int x = 0; x < gridWidth; ++x) {
                            for (unsigned int y = 0; y < gridWidth; ++y) {
                                flamegpu::AgentVector::Agent instance = init_pop[i++];
                                instance.setVariable<unsigned int, 2>("pos", { x, y });
                                // 10% chance of cell holding an agent
                                if (normal(rng) < pOccupation) {
                                    instance.setVariable<int>("agent_id", agent_id++);
                                    instance.setVariable<int>("status", AGENT_STATUS_OCCUPIED);
                                    instance.setVariable<int>("sugar_level", agent_sugar_dist(rng));
                                    instance.setVariable<int>("metabolism", agent_metabolism_dist(rng));
                                } else {
                                    instance.setVariable<int>("agent_id", -1);
                                    instance.setVariable<int>("status", AGENT_STATUS_UNOCCUPIED);
                                    instance.setVariable<int>("sugar_level", 0);
                                    instance.setVariable<int>("metabolism", 0);
                                }
                                // environment specific var
                                unsigned int env_sugar_lvl = 0;
                                const int hotspot_core_size = 5;
                                for (auto& hs : sugar_hotspots) {
                                    // Workout the highest sugar lvl from a nearby hotspot
                                    int hs_x = static_cast<int>(std::get<0>(hs));
                                    int hs_y = static_cast<int>(std::get<1>(hs));
                                    // distance to hotspot
                                    float hs_dist = static_cast<float>(sqrt(pow(hs_x - static_cast<int>(x), 2.0f) + pow(hs_y - static_cast<int>(y), 2.0f)));

                                    // four bands of sugar with increasing radius of 5
                                    env_sugar_lvl += 4 - std::min<int>(4, static_cast<int>(floor(hs_dist / hotspot_core_size)));
                                }
                                env_sugar_lvl = env_sugar_lvl > ENV_SUGAR_MAX_CAPACITY ? ENV_SUGAR_MAX_CAPACITY : env_sugar_lvl;
                                instance.setVariable<int>("env_max_sugar_level", env_sugar_lvl);  // All cells begin at their local max sugar
                                instance.setVariable<int>("env_sugar_level", env_sugar_lvl);
#ifdef VISUALISATION
                                // Redundant separate floating point position vars for vis
                                instance.setVariable<float>("x", static_cast<float>(x));
                                instance.setVariable<float>("y", static_cast<float>(y));
#endif
                            }
                        }
                        cudaSimulation.setPopulationData(init_pop);
                    }

                    /**
                     * Execution
                     */
#ifdef VISUALISATION
                    cudaSimulation.simulate();
                    visualisation.join();
#else

              
                     // Simulate and log for Histogram runs (output line per resolution step)
                    if (experiment.histogram) {
                        // reset mean unserveloved counters
                        std::fill(std::begin(mean_unresolved_count), std::end(mean_unresolved_count), 0);
                        // Iterate for number of steps and update the histogram representing the timings for number of resolution steps required
                        unsigned int sum_occupied_count = 0;
                        occupied = 0;  // reset the counter used in the exit condition
                        for (unsigned int i = 0; i < experiment.steps; i++) {
                            exit_condition_iterations = 0;  // reset the counter used in the exit condition
                            std::fill(std::begin(step_unresolved_count), std::end(step_unresolved_count), 0);
                            cudaSimulation.step();
                            // accumulate the number of unresolved at each step
                            for (unsigned int j = 0; j < 9; j++) {
                                mean_unresolved_count[j] += step_unresolved_count[j];
                            }
                            //accumulate the occupied count (i.e. numberof alive agents)
                            sum_occupied_count += occupied;
                        }
                        // average occpied count over steps
                        float mean_occupied = static_cast<float>(sum_occupied_count) / experiment.steps;
                        // average the iterations times and unresolved count and log
                        for (unsigned int i = 0; i < 9; i++) {
                            if (mean_unresolved_count[i])
                                mean_unresolved_count[i] /= experiment.steps;
                            // log histogram data to csv (repetition,grid_width,pop_size,resolution_iterations,average_s)
                            csv << repetition << "," << gridWidth << "," << popSize << "," << pOccupation << "," << mean_occupied << "," << i + 1 << "," << mean_unresolved_count[i] << std::endl;
                        }  
                    // Simulate and log for NON Histogram runs
                    } else {
                        unsigned int sum_pop_count = 0;
                        double sum_step_time = 0;
                        for (unsigned int i = 0; i < experiment.steps; i++) {
                            occupied = 0;  // reset the counter used in the exit condition
                            cudaSimulation.step();
                            unsigned int step_pop_count = occupied;
                            double step_time = cudaSimulation.getElapsedTimeStep(i);
                            // write step csv
                            csv_step << repetition << "," << gridWidth << "," << popSize << "," << pOccupation << "," << i << "," << step_time << "," << step_pop_count << std::endl;
                            // update average
                            sum_pop_count += step_pop_count;
                            sum_step_time += step_time;
                        }


                        // log total simulation time
                        const double averageStepTime = sum_step_time / static_cast<double>(BENCHMARK_STEPS);
                        const double averagePopCount = static_cast<double>(sum_pop_count) / static_cast<double>(BENCHMARK_STEPS);
                        // log timings to csv (repetition,grid_width,pop_size,s_step_mean)
                        csv << repetition << "," << gridWidth << "," << popSize << "," << pOccupation << "," << averageStepTime << "," << averagePopCount << std::endl;
                    }
#endif
                }
            }
        }
    }

    return 0;
}
